#include "hip/hip_runtime.h"
#include "app.cuh"
#include <glm/gtc/type_ptr.hpp>

// For GLFW stuffs
App *bound_app = nullptr;

App::App(YYLVVRes &res) : res(res),
    window(res.window),
    valid(false)
{
    if (!init())
    {
        std::cerr << "Failed to initialize UI & its resources?" << std::endl;
        return;
    }
    valid = true;
}

App::~App()
{
    if (render_state) {
        render_state->destroy();
    }
    CHECK_CUDA_ERROR(hipDestroyTextureObject(ctf_tex_cuda));
    CHECK_CUDA_ERROR(hipFreeArray(ctf_data_cuda));
}

bool App::init()
{
    std::cout << "Initializing bounding box and bounding box program." << std::endl;
    const glm::vec3 a = glm::vec3(-0.5f, -0.5f, -0.5f),
                b = glm::vec3(0.5f, -0.5f, -0.5f),
                c = glm::vec3(0.5f, -0.5f, 0.5f),
                d = glm::vec3(-0.5f, -0.5f, 0.5f),
                e = a + glm::vec3(0.0f, 1.0f, 0.0f),
                f = b + glm::vec3(0.0f, 1.0f, 0.0f),
                g = c + glm::vec3(0.0f, 1.0f, 0.0f),
                h = d + glm::vec3(0.0f, 1.0f, 0.0f);

    glm::vec3 bounding_box_data[] = 
    {
            a, b, b, c, c, d, d, a,
            e, f, f, g, g, h, h, e,
            a, b, b, f, f, e, e, a,
            d, c, c, g, g, h, h, d,
            a, d, d, h, h, e, e, a,
            b, c, c, g, g, f, f, b
    };

    bounding_box_vao = VAO::make_vao(bounding_box_data,
                                     sizeof(bounding_box_data),
                                     GL_STATIC_DRAW,
                                     {VertexAttribPointer(0,
                                                          3,
                                                          GL_FLOAT,
                                                          GL_FALSE,
                                                          sizeof(float) * 3,
                                                          nullptr)},
                                     GLDrawCall(GL_LINES, 0, 48));

    std::cout << "Compiling line drawing program." << std::endl;
    bounding_box_program = Program::make_program("shaders/lines.vert", "shaders/lines.frag");
    
    if (!bounding_box_program || !bounding_box_program->valid) 
    {
        std::cerr << "Cannot link line-drawing program?" << std::endl;
        return false;
    }

    std::cout << "Configuring OpenGL & GLFW." << std::endl;
    glEnable(GL_DEPTH_TEST);
    glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    glfwSetKeyCallback(window, key_callback_glfw);
    glfwSetCursorPosCallback(window, cursor_pos_callback_glfw);
    render_state = nullptr;
    glfwGetFramebufferSize(window, &screen_width, &screen_height);
    glfwSwapInterval(1);
    last_instant = glfwGetTime();
    delta_time = 0.0f;

    // Initialize delta wing
    std::cout << "Evaluating delta wing bounding box and allocating graphics resources." << std::endl;
    delta_wing_bounding_box = res.vf_tex.get_bounding_box();
    float x_min = 50.0f;
    float x_max = 169.0f;
    float y_min = 27.5f;
    float y_mid = 100.0f;
    float y_max = 172.5f;
    float z = 0.0f;
    std::vector<float3> delta_wing_fl3 = 
    {
        make_float3(x_min, y_mid, z),
        make_float3(x_max, y_min, z),
        make_float3(x_max, y_max, z)
    };
    std::cout << "Compiling delta wing shader." << std::endl;
    delta_wing_vao = VAO::make_vao(delta_wing_fl3.data(), delta_wing_fl3.size() * sizeof(float3), GL_STATIC_DRAW,
                                   {
                                       VertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 3, nullptr)
                                   },
                                   GLDrawCall(GL_TRIANGLES, 0, 3));
    delta_wing_program = Program::make_program("shaders/lines.vert", "shaders/delta.frag");
    
    if (!delta_wing_program || !delta_wing_program->valid) 
    {
        return false;
    }

    // Color transfer function (CTF)
    std::vector<float4> vector_magnitude_ctf;
    std::cout << "Initializing color transfer function for CUDA: creating device array." << std::endl;
    vector_magnitude_ctf.push_back(make_float4(0.4f, 0.6f, 0.9f, 1.0f)); // TODO: 1D texture is weird because it has a
    vector_magnitude_ctf.push_back(make_float4(0.0f, 1.0f, 0.0f, 1.0f)); //       weird padding of 0.5
    vector_magnitude_ctf.push_back(make_float4(0.9f, 0.9f, 0.0f, 1.0f));
    vector_magnitude_ctf.push_back(make_float4(1.0f, 0.0f, 0.0f, 1.0f));
    int vm_size_in_bytes = vector_magnitude_ctf.size() * sizeof(float4);
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    CHECK_CUDA_ERROR(hipMallocArray(&ctf_data_cuda, &desc, vector_magnitude_ctf.size(), 0, 0));
    CHECK_CUDA_ERROR(hipMemcpy2DToArray(ctf_data_cuda, 0, 0, vector_magnitude_ctf.data(), vm_size_in_bytes, vm_size_in_bytes, 1, hipMemcpyHostToDevice));

    std::cout << "Creating vector magnitude CTF texture." << std::endl;
    hipResourceDesc rdesc;
    std::memset(&rdesc, 0, sizeof(hipResourceDesc));
    {
        rdesc.resType = hipResourceTypeArray;
        rdesc.res.array.array = ctf_data_cuda;
    }
    hipTextureDesc tdesc;
    std::memset(&tdesc, 0, sizeof(hipTextureDesc));
    {
        tdesc.addressMode[0] = hipAddressModeWrap;
        tdesc.filterMode = hipFilterModeLinear;
        tdesc.readMode = hipReadModeElementType;
        tdesc.normalizedCoords = 0; // let's try normalizing it
    }
    CHECK_CUDA_ERROR(hipCreateTextureObject(&ctf_tex_cuda, &rdesc, &tdesc, nullptr));
    std::cout << "Vector magnitude CTF texture creation complete." << std::endl;

    align_camera();

    return true;
}

void App::key_callback_glfw(GLFWwindow *window, int key, int scancode, int action, int mods)
{
    assert(bound_app != nullptr);
    bound_app->key_callback(window, key, scancode, action, mods);
}

void App::cursor_pos_callback_glfw(GLFWwindow *window, double xpos, double ypos)
{
    assert(bound_app != nullptr);
    bound_app->cursor_pos_callback(window, xpos, ypos);
}

void App::key_callback(GLFWwindow *window, int key, int scancode, int action, int mods)
{
    if (action != 1) {
        return;
    }
    switch (key) {
        case GLFW_KEY_L:
            switch_state(std::make_shared<LineGlyphRenderState>());
            break;

        case GLFW_KEY_G:
            switch_state(std::make_shared<ArrowGlyphRenderState>());
            break;

        case GLFW_KEY_Z:
            switch_state(std::make_shared<StreamLineRenderState>());
            break;
    }
    if (render_state) {
        render_state->key_pressed(*this, key);
    }
}

void App::cursor_pos_callback(GLFWwindow *window, double xpos, double ypos)
{
    ypos = -ypos;
    xpos = -xpos;
    if (!camera.prev_cursor_pos) {
        camera.prev_cursor_pos = glm::dvec2(xpos, ypos);
        return;
    }
    glm::dvec2 curr_pos = glm::dvec2(xpos, ypos);
    glm::dvec2 delta_pos = curr_pos - *camera.prev_cursor_pos;
    camera.yaw += delta_pos.x * camera.sensitivity;
    camera.pitch += delta_pos.y * camera.sensitivity;
    camera.prev_cursor_pos = curr_pos;
    camera.update_components(screen_width, screen_height);
}

void App::align_camera()
{
    glm::vec3 extent = delta_wing_bounding_box.extend(); // TODO: a typo
    float max_ext = glm::max(glm::max(extent.x, extent.y), extent.z);
    float init_dist = glm::max(extent.x, extent.y) * 0.5f * sqrt(3.0f);
    camera.eye = delta_wing_bounding_box.center() - glm::vec3(0.0f, 0.0f, init_dist);
    camera.speed = max_ext * 0.1f; // whole thing in 10 seconds
    camera.z_near = 1.0f;
    camera.z_far = max_ext * 2.0f + fabs(init_dist);
    camera.update_components(screen_width, screen_height);
}

void App::handle_continuous_key_events()
{
    if (glfwGetKey(window, GLFW_KEY_W)) 
    {
        camera.eye += camera.front * camera.speed * delta_time;
        camera.update_components(screen_width, screen_height);
    }
    if (glfwGetKey(res.window, GLFW_KEY_S)) 
    {
        camera.eye -= camera.front * camera.speed * delta_time;
        camera.update_components(screen_width, screen_height);
    }
    if (glfwGetKey(res.window, GLFW_KEY_A)) 
    {
        camera.eye -= camera.right * camera.speed * delta_time;
        camera.update_components(screen_width, screen_height);
    }
    if (glfwGetKey(res.window, GLFW_KEY_D)) 
    {
        camera.eye += camera.right * camera.speed * delta_time;
        camera.update_components(screen_width, screen_height);
    }
}

void App::loop()
{
    while (!glfwWindowShouldClose(window)) 
    {
        bound_app = this;
        glfwPollEvents();

        double this_instant = glfwGetTime();
        delta_time = (float) (this_instant - last_instant);
        last_instant = this_instant;

        handle_continuous_key_events();

        if (render_state) 
        {
            render_state->process_events(*this);
        }

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        draw_delta_wing();

        if (render_state) 
        {
            render_state->render(*this);
        }

        glfwSwapBuffers(window);
    }
}

void App::draw_delta_wing() const
{
    // 1. Draw the bounding box (that we calculated)
    bounding_box_program->use();
    glm::mat4 model = glm::translate(glm::mat4(1.0f), delta_wing_bounding_box.center());
    model = glm::scale(model, delta_wing_bounding_box.extend());
    glUniformMatrix4fv(bounding_box_program->at("model"), 1, GL_FALSE, glm::value_ptr(model));
    glUniformMatrix4fv(bounding_box_program->at("view"), 1, GL_FALSE, glm::value_ptr(camera.view));
    glUniformMatrix4fv(bounding_box_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(camera.perspective));
    bounding_box_vao->draw();

    // 2. Draw the delta wing triangle
    delta_wing_program->use();
    glUniformMatrix4fv(delta_wing_program->at("model"), 1, GL_FALSE, glm::value_ptr(glm::mat4(1.0f)));
    glUniformMatrix4fv(delta_wing_program->at("view"), 1, GL_FALSE, glm::value_ptr(camera.view));
    glUniformMatrix4fv(delta_wing_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(camera.perspective));
    delta_wing_vao->draw();
}

void App::switch_state(std::shared_ptr<RenderState> new_state)
{
    if (render_state != nullptr) {
        render_state->destroy();
    }
    render_state = new_state;
    render_state->initialize(*this);
}
