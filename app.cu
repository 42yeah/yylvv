#include "hip/hip_runtime.h"
#include "app.cuh"
#include <glm/gtc/type_ptr.hpp>
#include <fstream>
#include <iomanip>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include "debug_kernels.cuh"

// For GLFW stuffs
App *bound_app = nullptr;

App::App(YYLVVRes &res) : res(res),
    window(res.window),
    valid(false),
    user_interface_mode(false),
    visualization_mode(0),
    should_draw_bounding_box(true),
    should_draw_delta_wing(true)
{
    if (!init())
    {
        std::cerr << "Failed to initialize UI & its resources?" << std::endl;
        return;
    }
    valid = true;
}

App::~App()
{
    if (render_state) 
    {
        render_state->destroy();
    }

    std::cout << "Destroying the rest of the app..." << std::endl;
    
    CHECK_CUDA_ERROR(hipDestroyTextureObject(ctf_tex_cuda));
    CHECK_CUDA_ERROR(hipFreeArray(ctf_data_cuda));

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
}

bool App::init()
{
    std::cout << "Initializing bounding box and bounding box program." << std::endl;
    const glm::vec3 a = glm::vec3(-0.5f, -0.5f, -0.5f),
                b = glm::vec3(0.5f, -0.5f, -0.5f),
                c = glm::vec3(0.5f, -0.5f, 0.5f),
                d = glm::vec3(-0.5f, -0.5f, 0.5f),
                e = a + glm::vec3(0.0f, 1.0f, 0.0f),
                f = b + glm::vec3(0.0f, 1.0f, 0.0f),
                g = c + glm::vec3(0.0f, 1.0f, 0.0f),
                h = d + glm::vec3(0.0f, 1.0f, 0.0f);

    glm::vec3 bounding_box_data[] = 
    {
            a, b, b, c, c, d, d, a,
            e, f, f, g, g, h, h, e,
            a, b, b, f, f, e, e, a,
            d, c, c, g, g, h, h, d,
            a, d, d, h, h, e, e, a,
            b, c, c, g, g, f, f, b
    };

    bounding_box_vao = VAO::make_vao(bounding_box_data,
                                     sizeof(bounding_box_data),
                                     GL_STATIC_DRAW,
                                     {VertexAttribPointer(0,
                                                          3,
                                                          GL_FLOAT,
                                                          GL_FALSE,
                                                          sizeof(float) * 3,
                                                          nullptr)},
                                     GLDrawCall(GL_LINES, 0, 48));

    std::cout << "Compiling line drawing program." << std::endl;
    bounding_box_program = Program::make_program("shaders/lines.vert", "shaders/lines.frag");
    
    if (!bounding_box_program || !bounding_box_program->valid) 
    {
        std::cerr << "Cannot link line-drawing program?" << std::endl;
        return false;
    }

    std::cout << "Configuring OpenGL & GLFW." << std::endl;
    glEnable(GL_DEPTH_TEST);
    glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    glfwSetKeyCallback(window, key_callback_glfw);
    glfwSetCursorPosCallback(window, cursor_pos_callback_glfw);
    render_state = nullptr;
    glfwGetFramebufferSize(window, &screen_width, &screen_height);
    glfwSwapInterval(1);
    last_instant = glfwGetTime();
    delta_time = 0.0f;

    // Initialize delta wing
    std::cout << "Evaluating delta wing bounding box and allocating graphics resources." << std::endl;
    delta_wing_bounding_box = res.vf_tex.get_bounding_box();
    float x_min = 50.0f;
    float x_max = 169.0f;
    float y_min = 27.5f;
    float y_mid = 100.0f;
    float y_max = 172.5f;
    float z = 0.0f;
    std::vector<float3> delta_wing_fl3 = 
    {
        make_float3(x_min, y_mid, z),
        make_float3(x_max, y_min, z),
        make_float3(x_max, y_max, z)
    };
    std::cout << "Compiling delta wing shader." << std::endl;
    delta_wing_vao = VAO::make_vao(delta_wing_fl3.data(), delta_wing_fl3.size() * sizeof(float3), GL_STATIC_DRAW,
                                   {
                                       VertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(float) * 3, nullptr)
                                   },
                                   GLDrawCall(GL_TRIANGLES, 0, 3));
    delta_wing_program = Program::make_program("shaders/lines.vert", "shaders/delta.frag");
    
    if (!delta_wing_program || !delta_wing_program->valid) 
    {
        return false;
    }

    // Color transfer function (CTF)
    std::vector<float4> vector_magnitude_ctf;
    std::cout << "Initializing color transfer function for CUDA: creating device array." << std::endl;
    vector_magnitude_ctf.push_back(make_float4(0.4f, 0.6f, 0.9f, 1.0f)); // TODO: 1D texture is weird because it has a
    vector_magnitude_ctf.push_back(make_float4(0.0f, 1.0f, 0.0f, 1.0f)); //       weird padding of 0.5
    vector_magnitude_ctf.push_back(make_float4(0.9f, 0.9f, 0.0f, 1.0f));
    vector_magnitude_ctf.push_back(make_float4(1.0f, 0.0f, 0.0f, 1.0f));
    int vm_size_in_bytes = vector_magnitude_ctf.size() * sizeof(float4);
    hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
    CHECK_CUDA_ERROR(hipMallocArray(&ctf_data_cuda, &desc, vector_magnitude_ctf.size(), 0, 0));
    CHECK_CUDA_ERROR(hipMemcpy2DToArray(ctf_data_cuda, 0, 0, vector_magnitude_ctf.data(), vm_size_in_bytes, vm_size_in_bytes, 1, hipMemcpyHostToDevice));

    std::cout << "Creating vector magnitude CTF texture." << std::endl;
    hipResourceDesc rdesc;
    std::memset(&rdesc, 0, sizeof(hipResourceDesc));
    {
        rdesc.resType = hipResourceTypeArray;
        rdesc.res.array.array = ctf_data_cuda;
    }
    hipTextureDesc tdesc;
    std::memset(&tdesc, 0, sizeof(hipTextureDesc));
    {
        tdesc.addressMode[0] = hipAddressModeWrap;
        tdesc.filterMode = hipFilterModeLinear;
        tdesc.readMode = hipReadModeElementType;
        tdesc.normalizedCoords = 0; // let's try normalizing it
    }
    CHECK_CUDA_ERROR(hipCreateTextureObject(&ctf_tex_cuda, &rdesc, &tdesc, nullptr));
    std::cout << "Vector magnitude CTF texture creation complete." << std::endl;

    align_camera();

    // Setup ImGui
    ImGui::CreateContext();
    ImGuiIO &io = ImGui::GetIO();
    io.IniFilename = nullptr;

    ImGui::StyleColorsDark();

    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 330 core");

    return true;
}

void App::key_callback_glfw(GLFWwindow *window, int key, int scancode, int action, int mods)
{
    assert(bound_app != nullptr);
    bound_app->key_callback(window, key, scancode, action, mods);
}

void App::cursor_pos_callback_glfw(GLFWwindow *window, double xpos, double ypos)
{
    assert(bound_app != nullptr);
    bound_app->cursor_pos_callback(window, xpos, ypos);
}

void App::key_callback(GLFWwindow *window, int key, int scancode, int action, int mods)
{
    if (action != 1) 
    {
        return;
    }

    if (user_interface_mode && key == GLFW_KEY_SPACE)
    {
        set_user_interface_mode(false);
        return;
    }

    switch (key) 
    {
        case GLFW_KEY_L:
            switch_state(std::make_shared<LineGlyphRenderState>());
            visualization_mode = 1;
            break;

        case GLFW_KEY_G:
            switch_state(std::make_shared<ArrowGlyphRenderState>());
            visualization_mode = 2;
            break;

        case GLFW_KEY_Z:
            switch_state(std::make_shared<StreamLineRenderState>());
            visualization_mode = 3;
            break;

        case GLFW_KEY_SPACE:
            set_user_interface_mode(!user_interface_mode);
            break;

        case GLFW_KEY_GRAVE_ACCENT:
            debug_vf();
            break;
    }

    if (render_state) 
    {
        render_state->key_pressed(*this, key);
    }
}

void App::cursor_pos_callback(GLFWwindow *window, double xpos, double ypos)
{
    ypos = -ypos;
    xpos = -xpos;
    
    if (!camera.prev_cursor_pos || user_interface_mode) 
    {
        camera.prev_cursor_pos = glm::dvec2(xpos, ypos);
        return;
    }

    glm::dvec2 curr_pos = glm::dvec2(xpos, ypos);
    glm::dvec2 delta_pos = curr_pos - *camera.prev_cursor_pos;
    camera.yaw += delta_pos.x * camera.sensitivity;
    camera.pitch += delta_pos.y * camera.sensitivity;
    camera.prev_cursor_pos = curr_pos;
    camera.update_components(screen_width, screen_height);
}

void App::align_camera()
{
    glm::vec3 extent = delta_wing_bounding_box.extend(); // TODO: a typo
    float max_ext = glm::max(glm::max(extent.x, extent.y), extent.z);
    float init_dist = glm::max(extent.x, extent.y) * 0.5f * sqrt(3.0f);
    camera.eye = delta_wing_bounding_box.center() - glm::vec3(0.0f, 0.0f, init_dist);
    camera.speed = max_ext * 0.1f; // whole thing in 10 seconds
    camera.z_near = 1.0f;
    camera.z_far = max_ext * 2.0f + fabs(init_dist);
    camera.update_components(screen_width, screen_height);
}

void App::handle_continuous_key_events()
{
    if (user_interface_mode)
    {
        return;
    }

    if (glfwGetKey(window, GLFW_KEY_W)) 
    {
        camera.eye += camera.front * camera.speed * delta_time;
    }
    if (glfwGetKey(res.window, GLFW_KEY_S)) 
    {
        camera.eye -= camera.front * camera.speed * delta_time;
    }
    if (glfwGetKey(res.window, GLFW_KEY_A)) 
    {
        camera.eye -= camera.right * camera.speed * delta_time;
    }
    if (glfwGetKey(res.window, GLFW_KEY_D)) 
    {
        camera.eye += camera.right * camera.speed * delta_time;
    }
}

void App::loop()
{
    while (!glfwWindowShouldClose(window)) 
    {
        if (!valid)
        {
            glfwSetWindowShouldClose(window, GLFW_TRUE);
        }

        bound_app = this;
        glfwPollEvents();

        double this_instant = glfwGetTime();
        delta_time = (float) (this_instant - last_instant);
        last_instant = this_instant;

        handle_continuous_key_events();

        if (render_state) 
        {
            render_state->process_events(*this);
        }

        glfwGetFramebufferSize(window, &screen_width, &screen_height);
        glViewport(0, 0, screen_width, screen_height);
        camera.update_components(screen_width, screen_height);

        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        draw_delta_wing();

        if (render_state) 
        {
            render_state->render(*this);
        }

        if (user_interface_mode)
        {
            ImGui_ImplOpenGL3_NewFrame();
            ImGui_ImplGlfw_NewFrame();
            ImGui::NewFrame();

            draw_user_controls();
            if (render_state)
            {
                render_state->draw_user_controls(*this);
            }

            ImGui::Render();
            ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        }

        glfwSwapBuffers(window);
    }
}

void App::draw_delta_wing() const
{
    if (should_draw_bounding_box)
    {
        // 1. Draw the bounding box (that we calculated)
        bounding_box_program->use();
        glm::mat4 model = glm::translate(glm::mat4(1.0f), delta_wing_bounding_box.center());
        model = glm::scale(model, delta_wing_bounding_box.extend());
        glUniformMatrix4fv(bounding_box_program->at("model"), 1, GL_FALSE, glm::value_ptr(model));
        glUniformMatrix4fv(bounding_box_program->at("view"), 1, GL_FALSE, glm::value_ptr(camera.view));
        glUniformMatrix4fv(bounding_box_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(camera.perspective));
        bounding_box_vao->draw();
    }

    if (should_draw_delta_wing)
    {
        // 2. Draw the delta wing triangle
        delta_wing_program->use();
        glUniformMatrix4fv(delta_wing_program->at("model"), 1, GL_FALSE, glm::value_ptr(glm::mat4(1.0f)));
        glUniformMatrix4fv(delta_wing_program->at("view"), 1, GL_FALSE, glm::value_ptr(camera.view));
        glUniformMatrix4fv(delta_wing_program->at("perspective"), 1, GL_FALSE, glm::value_ptr(camera.perspective));
        delta_wing_vao->draw();
    }
}

void App::switch_state(std::shared_ptr<RenderState> new_state)
{
    if (render_state != nullptr) 
    {
        render_state->destroy();
    }
    
    render_state = new_state;
    if (new_state)
    {
        render_state->initialize(*this);    
    }
}

void App::draw_user_controls()
{
    static std::vector<std::string> supported = 
    {
        "No visualizations",
        "Line glyphs",
        "Arrow glyphs",
        "Streamlines"
    };

    ImGui::SetNextWindowPos({0, 0}, ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowSize({220, 200}, ImGuiCond_FirstUseEver);
    if (ImGui::Begin("Visualizations"))
    {
        ImGui::Text("Visualization mode");
        ImGui::SetNextItemWidth(-FLT_MIN);
        if (ImGui::BeginListBox("Streamline Visualizations List"))
        {
            for (int i = 0; i < supported.size(); i++)
            {
                bool is_selected = visualization_mode == i;

                if (ImGui::Selectable(supported[i].c_str(), is_selected))
                {
                    visualization_mode = i;
                    switch (visualization_mode)
                    {
                        case 0:
                            switch_state(nullptr);
                            break;

                        case 1:
                            switch_state(std::make_shared<LineGlyphRenderState>());
                            break;

                        case 2:
                            switch_state(std::make_shared<ArrowGlyphRenderState>());
                            break;

                        case 3:
                            switch_state(std::make_shared<StreamLineRenderState>());
                            break;
                    }
                }

                if (is_selected)
                {
                    ImGui::SetItemDefaultFocus();
                }
            }
            ImGui::EndListBox();
        }
    }
    ImGui::End();

    ImGui::SetNextWindowPos({0, 200}, ImGuiCond_FirstUseEver);
    ImGui::SetNextWindowSize({250, 180}, ImGuiCond_FirstUseEver);
    if (ImGui::Begin("Camera configurations"))
    {
        // bool should_update_camera = false;
        float camera_eye[3] = {camera.eye.x, camera.eye.y, camera.eye.z};
        float camera_py[2] = {camera.pitch, camera.yaw};

        ImGui::Text("Camera position");
        ImGui::SetNextItemWidth(-FLT_MIN);
        ImGui::InputFloat3("##position", camera_eye);
        ImGui::Text("Pitch & yaw");
        ImGui::SetNextItemWidth(-FLT_MIN);
        ImGui::InputFloat2("##pitchyaw", camera_py);
        ImGui::Checkbox("Bounding box", &should_draw_bounding_box);
        ImGui::Checkbox("Delta wing", &should_draw_delta_wing);

        if (ImGui::Button("Favorite camera pose"))
        {
            favorite_camera_pose();
        }
        ImGui::SameLine();
        if (ImGui::Button("Restore"))
        {
            restore_camera_pose();
        }
    }
    ImGui::End();

    // ImGui::ShowDemoWindow();
}

void App::set_user_interface_mode(bool new_ui_mode)
{
    user_interface_mode = new_ui_mode;

    if (new_ui_mode)
    {
        // Stop capturing cursor and stop camera motion
        glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
    }
    else
    {
        glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    }
}

void App::favorite_camera_pose() const
{
    std::ofstream dumper("camera_pose.txt");
    if (!dumper.good())
    {
        std::cerr << "Bad writer for camera_pose.txt?" << std::endl;
        return;
    }

    constexpr auto max_precision = std::numeric_limits<long double>::digits10 + 1;
    dumper << std::setprecision(max_precision);
    dumper << camera.eye.x << " " << camera.eye.y << " " << camera.eye.z << std::endl;
    dumper << camera.pitch << " " << camera.yaw << " " << camera.sensitivity << " " << camera.speed << std::endl;
    dumper << camera.z_near << " " << camera.z_far << std::endl;
    dumper.close();
}

void App::restore_camera_pose()
{
    std::ifstream reader("camera_pose.txt");
    if (!reader.good())
    {
        std::cerr << "Cannot read camera_pose.txt?" << std::endl;
        return;
    }

    reader >> camera.eye.x >> camera.eye.y >> camera.eye.z
        >> camera.pitch >> camera.yaw >> camera.sensitivity >> camera.speed
        >> camera.z_near >> camera.z_far;
    reader.close();
    camera.update_components(screen_width, screen_height);    
}

void App::debug_vf() const
{
    std::cout << "Entering vector field debug mode." << std::endl;
    std::cout << "Input coordinates to sample texture." << std::endl;

    float x, y, z;
    
    while (true)
    {
        std::cin >> x >> y >> z;
        if (x < 0 || y < 0 || z < 0)
        {
            break;
        }

        float4 result = launch_sample_single_texture_3d_kernel(res.vf_tex.texture, x, y, z);
        std::cout << result.x << ", " << result.y << ", " << result.z << ", " << result.w << std::endl;
    }
}
