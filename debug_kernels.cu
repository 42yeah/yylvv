#include "hip/hip_runtime.h"
//
// Created by admin on 2022/9/21.
//

#include "debug_kernels.cuh"

__global__ void sample_single_texture_1d_kernel(hipTextureObject_t tex, float x, float4 *result) {
    *result = tex1D<float4>(tex, x);
}

float4 launch_sample_single_texture_1d_kernel(hipTextureObject_t tex, float x) {
    float4 *sampled = nullptr;
    hipMalloc((void **) &sampled, sizeof(float4));
    sample_single_texture_1d_kernel<<<1, 1>>>(tex, x, sampled);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // copy result back to host variable
    float4 res;
    hipMemcpy(&res, sampled, sizeof(float4), hipMemcpyDeviceToHost);
    hipFree(sampled);
    return res;
}

__global__ void
sample_single_texture_2d_kernel(hipTextureObject_t tex, float x, float y, float4 *result) {
    *result = tex2D<float4>(tex, x, y);
}

float4 launch_sample_single_texture_2d_kernel(hipTextureObject_t tex, float x, float y) {
    float4 *sampled = nullptr;
    hipMalloc((void **) &sampled, sizeof(float4));
    sample_single_texture_2d_kernel<<<1, 1>>>(tex, x, y, sampled);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // copy result back to host variable
    float4 res;
    hipMemcpy(&res, sampled, sizeof(float4), hipMemcpyDeviceToHost);
    hipFree(sampled);
    return res;
}

__global__ void sample_single_texture_3d_kernel(hipTextureObject_t tex, float x, float y, float z, float4 *result) {
    *result = tex3D<float4>(tex, x, y, z);
}

float4 launch_sample_single_texture_3d_kernel(hipTextureObject_t tex, float x, float y, float z) {
    float4 *sampled = nullptr;
    hipMalloc((void **) &sampled, sizeof(float4));
    sample_single_texture_3d_kernel<<<1, 1>>>(tex, x, y, z, sampled);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // copy result back to host variable
    float4 res;
    hipMemcpy(&res, sampled, sizeof(float4), hipMemcpyDeviceToHost);
    hipFree(sampled);
    return res;
}
